
#include <hip/hip_runtime.h>


__global__ void ImageWarpKernel(
	int pixel_count,
	const float* image, 
	const float* motion_vector, 
	const int batch,
	const int height,
	const int width,
	const int channel,
	float* output
) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int w = i % width;
	const int h = (i / width) % height;
	const int b = (i / (width * height)) % batch;

	// const int sx = 1;
	const int sy = width;
	const int sc = width * height;
	const int sb = channel * width * height;

	const int motion_vector_base = b*sc*2+h*sy*2+w*2;

	const float dx = motion_vector[motion_vector_base];
	const float dy = motion_vector[motion_vector_base+1];

	// (yy, xx) is the reprojected position
	const float yy = dy + h ;
	const float xx = dx + w;
	
	const int yy_int = round(yy);
	const int xx_int = round(xx);
	const float dyy = yy - yy_int;
	const float dxx = xx - xx_int;

	// xl, xr, yu, yb are the coordinates of the neibours near the reprojected position
	int xl = 0;
	int xr = 0;
	int yu = 0;
	int yb = 0;

	if (yy_int >= 0 && yy_int < height && xx_int >= 0 && xx_int < width)
	{
		if (dyy > 0) {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int;
				yb = yy_int+1;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int;
				yb = yy_int+1;
			}
		}
		else {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int-1;
				yb = yy_int;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int-1;
				yb = yy_int;
			}
		}
		
		const float u_diff    = abs(yy-yu);
		const float b_diff    = abs(yy-yb);
		const float l_diff    = abs(xx-xl);
		const float r_diff    = abs(xx-xr);
		const float ul_weight = 1-u_diff*l_diff;
		const float ur_weight = 1-u_diff*r_diff;
		const float bl_weight = 1-b_diff*l_diff;
		const float br_weight = 1-b_diff*r_diff;
		const float total     = ul_weight+ur_weight+bl_weight+br_weight;

		const int batch_part = b*sb;
		const int this_idx_base = batch_part+h*sy+w;
		
		if (yu >= 0 && yu < height && xl >= 0 && xl < width) {
			const float weight = ul_weight / total;
			const int   idx_base = batch_part + yu*sy + xl;
			for (int c = 0; c < channel; c++) {
				output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
			}
		}
		if (yu >= 0 && yu < height && xr >= 0 && xr < width) {
			const float weight = ur_weight / total;
			const int   idx_base = batch_part + yu*sy + xr;
			for (int c = 0; c < channel; c++) {
				output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
			}
		}
		if (yb >= 0 && yb < height && xl >= 0 && xl < width) {
			const float weight = bl_weight / total;
			const int   idx_base  = batch_part + yb*sy + xl;
			for (int c = 0; c < channel; c++) {
				output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
			}
		}
		if (yb >= 0 && yb < height && xr >= 0 && xr < width) {
			const float weight = br_weight / total;
			const int   idx_base  = batch_part + yb*sy + xr;
			for (int c = 0; c < channel; c++) {
				output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
			}
		}
	}
	
}

__global__ void ImageWarpAccumKernel(
	int pixel_count,
	const float* image, 
	const float* motion_vector, 
	const int batch,
	const int height,
	const int width,
	const int channel,
	const float alpha,
	float* output
) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int w = i % width;
	const int h = (i / width) % height;
	const int b = (i / (width * height)) % batch;

	// const int sx = 1;
	const int sy = width;
	const int sc = width * height;
	const int sb = channel * width * height;

	const int motion_vector_base = b*sc*2+h*sy*2+w*2;

	const float dx = motion_vector[motion_vector_base];
	const float dy = motion_vector[motion_vector_base+1];

	// (yy, xx) is the reprojected position
	const float yy = dy + h ;
	const float xx = dx + w;
	
	const int yy_int = round(yy);
	const int xx_int = round(xx);
	const float dyy = yy - yy_int;
	const float dxx = xx - xx_int;

	// xl, xr, yu, yb are the coordinates of the neibours near the reprojected position
	int xl = 0;
	int xr = 0;
	int yu = 0;
	int yb = 0;

	if (yy_int >= 0 && yy_int < height && xx_int >= 0 && xx_int < width)
	{
		if (dyy > 0) {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int;
				yb = yy_int+1;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int;
				yb = yy_int+1;
			}
		}
		else {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int-1;
				yb = yy_int;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int-1;
				yb = yy_int;
			}
		}
		
		const float u_diff    = abs(yy-yu);
		const float b_diff    = abs(yy-yb);
		const float l_diff    = abs(xx-xl);
		const float r_diff    = abs(xx-xr);
		const float ul_weight = 1-u_diff*l_diff;
		const float ur_weight = 1-u_diff*r_diff;
		const float bl_weight = 1-b_diff*l_diff;
		const float br_weight = 1-b_diff*r_diff;
		const float total     = ul_weight+ur_weight+bl_weight+br_weight;

		const int batch_part = b*sb;
		const int this_idx_base = batch_part+h*sy+w;

		float color[64] = {0};
		if (yu >= 0 && yu < height && xl >= 0 && xl < width) {
			const float weight = ul_weight / total;
			const int   idx_base = batch_part + yu*sy + xl;
			for (int c = 0; c < channel; c++) {
				color[c] += image[idx_base+c*sc] * weight;
			}
		}
		if (yu >= 0 && yu < height && xr >= 0 && xr < width) {
			const float weight = ur_weight / total;
			const int   idx_base = batch_part + yu*sy + xr;
			for (int c = 0; c < channel; c++) {
				color[c] += image[idx_base+c*sc] * weight;
			}
		}
		if (yb >= 0 && yb < height && xl >= 0 && xl < width) {
			const float weight = bl_weight / total;
			const int   idx_base  = batch_part + yb*sy + xl;
			for (int c = 0; c < channel; c++) {
				color[c] += image[idx_base+c*sc] * weight;
			}
		}
		if (yb >= 0 && yb < height && xr >= 0 && xr < width) {
			const float weight = br_weight / total;
			const int   idx_base  = batch_part + yb*sy + xr;
			for (int c = 0; c < channel; c++) {
				color[c] += image[idx_base+c*sc] * weight;
			}
		}
		for (int c = 0; c < channel; c++) {
			output[this_idx_base+c*sc] = output[this_idx_base+c*sc] * (1-alpha) + alpha * color[c];
		}
	}
}

__global__ void ImageWarpGradKernel(
	int pixel_count,
	const float* image, 
	const float* motion_vector, 
	const float* backprop, 
	const int batch,
	const int height,
	const int width,
	const int channel,
	float* output
) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int w = i % width;
	const int h = (i / width) % height;
	const int b = (i / (width * height)) % batch;

	// const int sx = 1;
	const int sy = width;
	const int sc = width * height;
	const int sb = channel * width * height;

	const int motion_vector_base = b*sc*2+h*sy*2+w*2;

	const float dx = motion_vector[motion_vector_base];
	const float dy = motion_vector[motion_vector_base+1];

	// (yy, xx) is the reprojected position
	const float yy = dy + h ;
	const float xx = dx + w;
	
	const int yy_int = round(yy);
	const int xx_int = round(xx);
	const float dyy = yy - yy_int;
	const float dxx = xx - xx_int;

	// xl, xr, yu, yb are the coordinates of the neibours near the reprojected position
	int xl = 0;
	int xr = 0;
	int yu = 0;
	int yb = 0;

	if (yy_int >= 0 && yy_int < height && xx_int >= 0 && xx_int < width)
	{
		if (dyy > 0) {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int;
				yb = yy_int+1;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int;
				yb = yy_int+1;
			}
		}
		else {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int-1;
				yb = yy_int;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int-1;
				yb = yy_int;
			}
		}
		
		const float u_diff    = abs(yy-yu);
		const float b_diff    = abs(yy-yb);
		const float l_diff    = abs(xx-xl);
		const float r_diff    = abs(xx-xr);
		const float ul_weight = 1-u_diff*l_diff;
		const float ur_weight = 1-u_diff*r_diff;
		const float bl_weight = 1-b_diff*l_diff;
		const float br_weight = 1-b_diff*r_diff;
		const float total     = ul_weight+ur_weight+bl_weight+br_weight;

		const int batch_part = b*sb;
		const int this_idx_base = batch_part+h*sy+w;
		
		if (yu >= 0 && yu < height && xl >= 0 && xl < width) {
			const float weight = ul_weight / total;
			const int   idx_base = batch_part + yu*sy + xl;
			for (int c = 0; c < channel; c++) {
				// output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
				output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight;
			}
		}
		if (yu >= 0 && yu < height && xr >= 0 && xr < width) {
			const float weight = ur_weight / total;
			const int   idx_base = batch_part + yu*sy + xr;
			for (int c = 0; c < channel; c++) {
				// output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
				output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight;
			}
		}
		if (yb >= 0 && yb < height && xl >= 0 && xl < width) {
			const float weight = bl_weight / total;
			const int   idx_base  = batch_part + yb*sy + xl;
			for (int c = 0; c < channel; c++) {
				// output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
				output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight;
			}
		}
		if (yb >= 0 && yb < height && xr >= 0 && xr < width) {
			const float weight = br_weight / total;
			const int   idx_base  = batch_part + yb*sy + xr;
			for (int c = 0; c < channel; c++) {
				// output[this_idx_base+c*sc] += image[idx_base+c*sc] * weight;
				output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight;
			}
		}
	}	
}

__global__ void ImageWarpAccumGradKernel(
	int pixel_count,
	const float* image, 
	const float* motion_vector, 
	const float* backprop, 
	const int batch,
	const int height,
	const int width,
	const int channel,
	const float alpha,
	float* img_grad_output,
	float* imgdst_grad_output
) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int w = i % width;
	const int h = (i / width) % height;
	const int b = (i / (width * height)) % batch;

	// const int sx = 1;
	const int sy = width;
	const int sc = width * height;
	const int sb = channel * width * height;

	const int motion_vector_base = b*sc*2+h*sy*2+w*2;

	const float dx = motion_vector[motion_vector_base];
	const float dy = motion_vector[motion_vector_base+1];

	// (yy, xx) is the reprojected position
	const float yy = dy + h ;
	const float xx = dx + w;
	
	const int yy_int = round(yy);
	const int xx_int = round(xx);
	const float dyy = yy - yy_int;
	const float dxx = xx - xx_int;

	// xl, xr, yu, yb are the coordinates of the neibours near the reprojected position
	int xl = 0;
	int xr = 0;
	int yu = 0;
	int yb = 0;

	const int batch_part = b*sb;
	const int this_idx_base = batch_part+h*sy+w;

	if (yy_int >= 0 && yy_int < height && xx_int >= 0 && xx_int < width)
	{
		if (dyy > 0) {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int;
				yb = yy_int+1;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int;
				yb = yy_int+1;
			}
		}
		else {
			if (dxx > 0) {
				xl = xx_int;
				xr = xx_int+1;
				yu = yy_int-1;
				yb = yy_int;
			}
			else {
				xl = xx_int-1;
				xr = xx_int;
				yu = yy_int-1;
				yb = yy_int;
			}
		}
		
		const float u_diff    = abs(yy-yu);
		const float b_diff    = abs(yy-yb);
		const float l_diff    = abs(xx-xl);
		const float r_diff    = abs(xx-xr);
		const float ul_weight = 1-u_diff*l_diff;
		const float ur_weight = 1-u_diff*r_diff;
		const float bl_weight = 1-b_diff*l_diff;
		const float br_weight = 1-b_diff*r_diff;
		const float total     = ul_weight+ur_weight+bl_weight+br_weight;

		if (yu >= 0 && yu < height && xl >= 0 && xl < width) {
			const float weight = ul_weight / total;
			const int   idx_base = batch_part + yu*sy + xl;
			for (int c = 0; c < channel; c++) {
				// color[c] += image[idx_base+c*sc] * weight;
				img_grad_output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight * alpha;
			}
		}
		if (yu >= 0 && yu < height && xr >= 0 && xr < width) {
			const float weight = ur_weight / total;
			const int   idx_base = batch_part + yu*sy + xr;
			for (int c = 0; c < channel; c++) {
				// color[c] += image[idx_base+c*sc] * weight;
				img_grad_output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight * alpha;
			}
		}
		if (yb >= 0 && yb < height && xl >= 0 && xl < width) {
			const float weight = bl_weight / total;
			const int   idx_base  = batch_part + yb*sy + xl;
			for (int c = 0; c < channel; c++) {
				// color[c] += image[idx_base+c*sc] * weight;
				img_grad_output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight * alpha;
			}
		}
		if (yb >= 0 && yb < height && xr >= 0 && xr < width) {
			const float weight = br_weight / total;
			const int   idx_base  = batch_part + yb*sy + xr;
			for (int c = 0; c < channel; c++) {
				// color[c] += image[idx_base+c*sc] * weight;
				img_grad_output[idx_base+c*sc] += backprop[this_idx_base+c*sc] * weight * alpha;
			}
		}
		for (int c = 0; c < channel; c++) {
			imgdst_grad_output[this_idx_base+c*sc] = backprop[this_idx_base+c*sc] * (1-alpha);
		}
	}
	else {
		for (int c = 0; c < channel; c++) {
			imgdst_grad_output[this_idx_base+c*sc] = backprop[this_idx_base+c*sc];
		}
	}
}


void ImageWarpKernelLauncher(
	const float* image,
	const float* motion_vector,
	const int* image_size,
	float* output
) {
	int batch = image_size[0];
	int channel = image_size[1];
	int height = image_size[2];
	int width = image_size[3];

	int pixel_count = batch * height * width;
	if (pixel_count > 0) {
		dim3 GRID_SIZE((pixel_count + 1023) / 1024);
		dim3 BLOCK_SIZE(1024);
		ImageWarpKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
			pixel_count,
			image,
			motion_vector,
			batch,
			height,
			width,
			channel,
			output);
	}
}

void ImageWarpAccumKernelLauncher(
	const float* image,
	const float* motion_vector,
	const int* image_size,
	const float alpha,
	float* output
) {
	int batch = image_size[0];
	int channel = image_size[1];
	int height = image_size[2];
	int width = image_size[3];

	int pixel_count = batch * height * width;
	if (pixel_count > 0) {
		dim3 GRID_SIZE((pixel_count + 1023) / 1024);
		dim3 BLOCK_SIZE(1024);
		ImageWarpAccumKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
			pixel_count,
			image,
			motion_vector,
			batch,
			height,
			width,
			channel,
			alpha,
			output);
	}
}
void ImageWarpGradKernelLauncher(
	const float* image,
	const float* motion_vector,
	const float* backprop,
	const int* image_size,
	float* output
) {
	int batch = image_size[0];
	int channel = image_size[1];
	int height = image_size[2];
	int width = image_size[3];

	int pixel_count = batch * height * width;
	if (pixel_count > 0) {
		dim3 GRID_SIZE((pixel_count + 1023) / 1024);
		dim3 BLOCK_SIZE(1024);
		ImageWarpGradKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
			pixel_count,
			image,
			motion_vector,
			backprop,
			batch,
			height,
			width,
			channel,
			output);
	}
}

void ImageWarpAccumGradKernelLauncher(
	const float* image,
	const float* motion_vector,
	const float* backprop,
	const int* image_size,
	const float alpha,
	float* img_grad_output,
	float* imgdst_grad_output
) {
	int batch = image_size[0];
	int channel = image_size[1];
	int height = image_size[2];
	int width = image_size[3];

	int pixel_count = batch * height * width;
	if (pixel_count > 0) {
		dim3 GRID_SIZE((pixel_count + 1023) / 1024);
		dim3 BLOCK_SIZE(1024);
		ImageWarpAccumGradKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
			pixel_count,
			image,
			motion_vector,
			backprop,
			batch,
			height,
			width,
			channel,
			alpha,
			img_grad_output,
			imgdst_grad_output);
	}
}